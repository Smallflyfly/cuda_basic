#include "hip/hip_runtime.h"
//
// Created by smallflyfly on 2021/5/17.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cuda_code.h"


void initData(float *a, int length) {
    for (int i=0; i<length; i++) {
        a[i] = i * 1.0;
    }
}

__global__ void sumCuda(const float *a, const float *b, float *sum_d) {
    int i = threadIdx.x;
    sum_d[i] = a[i] + b[i];
}

int main(int argc, char **argv) {
    int device = 0;
    hipSetDevice(device);

    int numElement = 10;

    int nBytes = sizeof(float) * numElement;

    float *a_h = (float*)malloc(nBytes);
    float *b_h = (float*)malloc(nBytes);
    float *sum_h = (float*) malloc(nBytes);

    memset(a_h, 0 , nBytes);
    memset(b_h, 0, nBytes);

    float *a_d, *b_d, *sum_d;
    CHECK(hipMalloc((float**)&a_d, nBytes));
    CHECK(hipMalloc((float**)&b_d, nBytes));
    CHECK(hipMalloc((float**)&sum_d, nBytes));

    initData(a_h, numElement);
    initData(b_h, numElement);

    CHECK(hipMemcpyAsync(a_d, a_h, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpyAsync(b_d, b_h, nBytes, hipMemcpyHostToDevice));

    dim3 block(numElement);
    dim3 grid(numElement / block.x);

    sumCuda<<<grid, block>>>(a_d, b_d, sum_d);

    CHECK(hipMemcpyAsync(sum_h, sum_d, nBytes, hipMemcpyDeviceToHost));

    for (int i=0; i<numElement; i++) {
        printf("%f\n", sum_h[i]);
    }

    hipDeviceReset();

    hipFree(a_d);
    hipFree(b_d);
    hipFree(sum_d);

    free(a_h);
    free(b_h);
    free(sum_h);

    return 0;
}