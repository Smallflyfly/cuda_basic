#include "hip/hip_runtime.h"
//
// Created by smallflyfly on 2021/6/3.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cuda_code.h"


__global__ void matMultiply(float *a, float *b, float *c, int width, int height) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    if (tx >= width || ty >= height) return;

    float mulValue = 0.0;
    for (int i = 0; i < width; i++) {
        mulValue += a[ty * width + i] * b[tx * height + i];
    }
    c[ty * width + tx] = mulValue;
}


int main() {
    int width = 1 << 2;
    int height = 1 << 2;

    float *ah, *bh, *ch;
    unsigned nBytes = width * height * sizeof(float);

    ah = (float*)malloc(nBytes);
    bh = (float*)malloc(nBytes);
    ch = (float*)malloc(nBytes);

    for (int i = 0; i < width * height; i++) {
        ah[i] = 1.0;
        bh[i] = 2.0;
    }

    float *ad, *bd, *cd;

    // malloc device
    CHECK(hipMalloc((void**)&ad, nBytes));
    CHECK(hipMalloc((void**)&bd, nBytes));
    CHECK(hipMalloc((void**)&cd, nBytes));

    // copy host data to device
    CHECK(hipMemcpyAsync(ad, ah, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpyAsync(bd, bh, nBytes, hipMemcpyHostToDevice));

    // block grid
    dim3 blockSize(2, 4);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // run kernel
    matMultiply<<<gridSize, blockSize>>>(ad, bd, cd, width, height);
    hipDeviceSynchronize();

    // copy result from device to host
    CHECK(hipMemcpyAsync(ch, cd, nBytes, hipMemcpyDeviceToHost));

    float maxError = 0.0;
    for (int i = 0; i < width * height; i++) {
        printf("%.2f ", ch[i]);
        if ((i+1) % width == 0) printf("\n");
    }


    // 释放内存

    hipFree(ad);
    hipFree(bd);
    hipFree(cd);

    free(ah);
    free(bh);
    free(ch);

    hipDeviceReset();

    return 0;
}